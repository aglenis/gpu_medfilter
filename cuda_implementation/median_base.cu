#include "hip/hip_runtime.h"
#include <stdio.h>

template <typename T,unsigned int WINDOW_SIZE>
__global__
void MedianFilter2D( T *input,T* output,int widthImage, int heightImage)
{
    int filter_offset=WINDOW_SIZE/2;
//y and x are oposite the cuda programming model
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int x = blockIdx.y * blockDim.y + threadIdx.y;
    if(y>heightImage || x>widthImage)
        return;

    T window[WINDOW_SIZE*WINDOW_SIZE];
    for (int counter=0; counter<WINDOW_SIZE*WINDOW_SIZE; counter++)
    {
        window[counter]=0;
    }
    int count=0;
    for( int k=y-filter_offset; k<=y+filter_offset; k++)
    {
        for (int l=x-filter_offset; l<=x+filter_offset; l++)
        {
	    if(k>=0 && l>=0 && k<heightImage && l<widthImage)
	      window[count++]=input[(k)*widthImage+(l)];

        }
    }
    insertionSort(window,WINDOW_SIZE*WINDOW_SIZE);

    output[y*widthImage + x]=window[WINDOW_SIZE*WINDOW_SIZE/2];

}

template <typename T,unsigned int WINDOW_SIZE>
__global__
void MedianFilter2D_histogram( T *input,T* output,int widthImage, int heightImage)
{
    int filter_offset=WINDOW_SIZE/2;
//y and x are oposite the cuda programming model
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int x = blockIdx.y * blockDim.y + threadIdx.y;
    if(y>heightImage || x>widthImage)
        return;

    T window[WINDOW_SIZE*WINDOW_SIZE];
    for (int counter=0; counter<WINDOW_SIZE*WINDOW_SIZE; counter++)
    {
        window[counter]=0;
    }
    int count=0;
    for( int k=max(y-filter_offset,0); k<=min(y+filter_offset,heightImage-1); k++)
    {
        for (int l=max(x-filter_offset,0); l<=min(x+filter_offset,widthImage-1); l++)
        {

            window[count++]=input[(k)*widthImage+(l)];

        }
    }
    unsigned int histogram_values[256];
    for( int i=0; i<256; i++)
    {
        histogram_values[i]=0;
    }
    for(int i=0; i<WINDOW_SIZE*WINDOW_SIZE; i++)
    {
        histogram_values[window[i]]++;
    }
    unsigned int target_value=(WINDOW_SIZE*WINDOW_SIZE-1)/2;
    unsigned int curr_sum=0;
    unsigned int curr_count=0;
    for( curr_count=0; curr_count<256; curr_count++)
    {
        curr_sum+=histogram_values[curr_count];
        if(curr_sum>target_value)
        {
            break;
        }
    }

    output[y*widthImage + x]=curr_count;

}

template <typename T,unsigned int WINDOW_SIZE>
__global__
void MedianFilter2D_partial( T *input,T* output,int widthImage, int heightImage)
{
    int filter_offset=WINDOW_SIZE/2;
//y and x are oposite the cuda programming model
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int x = blockIdx.y * blockDim.y + threadIdx.y;
    if(y>heightImage || x>widthImage)
        return;

    T window[WINDOW_SIZE*WINDOW_SIZE];
    for (int counter=0; counter<WINDOW_SIZE*WINDOW_SIZE; counter++)
    {
        window[counter]=0;
    }
    int count=0;
    for( int k=max(y-filter_offset,0); k<=min(y+filter_offset,heightImage-1); k++)
    {
        for (int l=max(x-filter_offset,0); l<=min(x+filter_offset,widthImage-1); l++)
        {

            window[count++]=input[(k)*widthImage+(l)];

        }
    }
    partialSelection(window,WINDOW_SIZE);

    output[y*widthImage + x]=window[WINDOW_SIZE*WINDOW_SIZE/2];

}

template <typename T,unsigned int WINDOW_SIZE>
__global__
void MedianFilter2D_forgetful( T *input,T* output,int widthImage, int heightImage)
{
    int filter_offset=WINDOW_SIZE/2;
//y and x are oposite the cuda programming model
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int x = blockIdx.y * blockDim.y + threadIdx.y;
    if(y>heightImage || x>widthImage)
        return;

    T window[WINDOW_SIZE*WINDOW_SIZE];
    for (int counter=0; counter<WINDOW_SIZE*WINDOW_SIZE; counter++)
    {
        window[counter]=0;
    }
    int count=0;
    for( int k=max(y-filter_offset,0); k<=min(y+filter_offset,heightImage-1); k++)
    {
        for (int l=max(x-filter_offset,0); l<=min(x+filter_offset,widthImage-1); l++)
        {

            window[count++]=input[(k)*widthImage+(l)];

        }
    }
    forgetfulSelection(window,WINDOW_SIZE);

    output[y*widthImage + x]=window[WINDOW_SIZE*WINDOW_SIZE/2];

}

__device__ void insertionSort(unsigned char window[],int size)
{
    int i , j;
    unsigned char temp;
    for(i = 0; i < size; i++) {
        temp = window[i];
        for(j = i-1; j >= 0 && temp < window[j]; j--) {
            window[j+1] = window[j];
        }
        window[j+1] = temp;
    }
}

__device__ void partialSelection(unsigned char * window,int size)
{
    // Order elements (only half of them)WINDOW_SIZE
    //TODO this works with a odd window size to avoid a ceil function
    for (unsigned int j=0; j<(size*size+1)/2; ++j)
    {
        // Find position of minimum element
        int min_index=j;
        for (unsigned int l=j+1; l<size*size; ++l)
            if (window[l] < window[min_index])
                min_index=l;

        // Put found minimum element in its place
        const unsigned char temp=window[j];
        window[j]=window[min_index];
        window[min_index]=temp;
    }
}

__device__ void swap_elements(unsigned char* array,int pos1,int pos2)
{
    unsigned char temp=array[pos1];
    array[pos1]=array[pos2];
    array[pos2]=temp;
}

__device__ void extrema_identification(unsigned char * window,int start_offset,int size)
{

    //identify the minimum and maximum elements in the array
    unsigned int min_index,max_index;
    min_index=max_index=start_offset;
    unsigned char max_value=window[start_offset];
    unsigned char min_value=window[start_offset];
    for( int i=start_offset+1; i<start_offset+size; i++)
    {
        if(window[i]<min_value)
        {
            min_index=i;
            min_value=window[i];
        }
        if(window[i]>max_value)
        {
            max_index=i;
            max_value=window[i];
        }

    }
    swap_elements(window,min_index,start_offset);
    swap_elements(window,max_index,size-1+start_offset);
}
__device__
void extrema_identification2(unsigned char * window,int start_offset,int size)
{

    //identify the minimum and maximum elements in the array
    unsigned int min_index,max_index;
    min_index=max_index=start_offset;
   
    unsigned char min_value=window[start_offset];
    for( int i=start_offset+1; i<start_offset+size; i++)
    {
        if(window[i]<min_value)
        {
            min_index=i;
            min_value=window[i];
        }


    }

   
    swap_elements(window,min_index,start_offset);
    

     unsigned char max_value=window[start_offset];
   for( int i=start_offset+1; i<start_offset+size; i++)
    {
        if(window[i]>max_value)
        {
            max_index=i;
            max_value=window[i];
        }
    }
    swap_elements(window,max_index,size-1+start_offset);

}

__device__ void forgetfulSelection(unsigned char * window,int size)
{
    int Rn=ceil((float)(size*size)/2)+1;
    extrema_identification2(window,0,Rn);
    int stop_nr=size*size-Rn;
    for (int step=1; step<=(stop_nr); step++)
    {
        window[Rn-1]=window[Rn+step-1];
        extrema_identification2(window,step,Rn-step);
    }
}


__device__
void initialize_histogram(unsigned int *histogram_array,unsigned char *image,int targetHeight,int targetWidth,int widthImage)
{
  int counter=0;
  for( int i=0;i<targetHeight;i++)
  {
    for(int j=0;j<targetWidth;j++)
    {
      histogram_array[image[i*widthImage+j]]++;
    }
  }
}
__device__
void remove_element_from_histogram(unsigned int * histogram_array,unsigned char * array,int heightPos,int widthPos,int heightImage,int widthImage)
{
  histogram_array[array[heightPos*widthImage+widthPos]]--;
}

__device__
void add_element_to_histogram(unsigned int * histogram_array,unsigned char * array,int heightPos,int widthPos,int heightImage,int widthImage)
{
  histogram_array[array[heightPos*widthImage+widthPos]]++;
}
__device__
unsigned char compute_mean_from_histogram(unsigned int * histogram_array,int window_size)
{
  unsigned int target_value=(window_size*window_size-1)/2;
        unsigned int curr_sum=0;
	int i=0;
		for( i=0;i<256;i++)
	{
	  curr_sum+=histogram_array[i];
	  if(curr_sum>target_value)
	  {break;}
	}
	return (unsigned char)i;
}

template <typename T,unsigned int WINDOW_SIZE>
__global__ void MedianFilter2D_histogram_fast(T * array,T * out_array,int heightImage,int widthImage)
{
  
//   unsigned int target_value=(WINDOW_SIZE*WINDOW_SIZE-1)/2;
  int r=WINDOW_SIZE/2;
//   printf("r is %d \n",r);
  unsigned int histogram_array[256];
  
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    for(int i=r;i<(heightImage-r);i++)
  {
      for(int m=0;m<256;m++){histogram_array[m]=0;}
  initialize_histogram(histogram_array,array+(i-r)*widthImage,WINDOW_SIZE,WINDOW_SIZE,widthImage);
  
    for(int j=r;j<(widthImage-r-1);j++)
    {
      T mean_value=compute_mean_from_histogram(histogram_array,WINDOW_SIZE);
      out_array[i*widthImage+j]=mean_value;
                  for(int k=-r;k<=r;k++)
      {
	
//   	if((j+r)!=(widthImage-r-1))
//   	{
	remove_element_from_histogram(histogram_array,array,i+k,j-1,heightImage,widthImage);
//   	printf("To compute element %d %d i am removing element %d %d \n",i,j,i+k,j-1);
 	add_element_to_histogram(histogram_array,array,i+k,j+2,heightImage,widthImage);
//  	printf("To compute element %d %d i am adding element %d %d \n",i,j,i+k,j+2);
//   	}
      }
      
//        unsigned char mean_value=compute_mean_from_histogram(histogram_array,window_size);
//       out_array[i*widthImage+j]=mean_value;

    }
  }
}

__global__
void histogram2d(unsigned char * array,unsigned char * out_array,unsigned int * histogram_array,int heightImage,int widthImage,int window_size)
{
  
//   unsigned int target_value=(WINDOW_SIZE*WINDOW_SIZE-1)/2;
  int r=window_size/2;
//   printf("r is %d \n",r);
//   unsigned int histogram_array[256];
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=r && i<(heightImage-r))
  {
       for(int m=0;m<256;m++){histogram_array[256*i+m]=0;}
  initialize_histogram(&histogram_array[256*i],array+(i-r)*widthImage,window_size,window_size,widthImage);
  
    for(int j=r;j<(widthImage-r-1);j++)
    {
      unsigned char mean_value=compute_mean_from_histogram(&histogram_array[256*i],window_size);
      out_array[i*widthImage+j]=mean_value;
                  for(int k=-r;k<=r;k++)
      {
	
//   	if((j+r)!=(widthImage-r-1))
//   	{
	remove_element_from_histogram(&histogram_array[256*i],array,i+k,j-1,heightImage,widthImage);
//   	printf("To compute element %d %d i am removing element %d %d \n",i,j,i+k,j-1);
 	add_element_to_histogram(&histogram_array[256*i],array,i+k,j+2,heightImage,widthImage);
//  	printf("To compute element %d %d i am adding element %d %d \n",i,j,i+k,j+2);
//   	}
      }
      
//        unsigned char mean_value=compute_mean_from_histogram(histogram_array,window_size);
//       out_array[i*widthImage+j]=mean_value;

    }
  }
}

template <typename T>
void callMedianFilter(T *input,T* output,unsigned int *d_histogram,int widthImage, int heightImage,int window_size, int threadsX, int threadsY,int implentation)
{
    dim3 block(threadsX, threadsY, 1);
    dim3 grid((int)ceil((float)heightImage / block.x),(int) (ceil((float)widthImage / block.y)), 1);
//     printf("original image is size %d %d gridx is %d grid y is %d \n",heightImage,widthImage,grid.x,grid.y);
//     	  int total_threads=threadsX*threadsY; 
	  int total_threads=256;
	  int total_blocks=ceil(heightImage/total_threads);
    switch(window_size)
    {
    case 3:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,3><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,3><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,3><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,3><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:

// 	  printf("total threads are %d total blocks are %d \n",total_threads,total_blocks);
// 	  hipMemset(d_histogram,0,256*total_threads);
// 	  (unsigned char * array,unsigned char * out_array,unsigned int * histogram_array,int heightImage,int widthImage,int window_size)
            histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
    case 5:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,5><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,5><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,5><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,5><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
    case 7:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,7><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,7><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,7><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,7><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
            histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
    case 9:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,9><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,9><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,9><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,9><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
	
	    case 15:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,15><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,15><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,15><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,15><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
	
		    case 17:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,17><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,17><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,17><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,17><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
	
			    case 25:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,25><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,25><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,25><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,25><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
	
				    case 31:
        switch(implentation) {
        case 1:
            MedianFilter2D<T,31><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 2:
            MedianFilter2D_histogram<T,31><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 3:
            MedianFilter2D_partial<T,31><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
        case 4:
            MedianFilter2D_forgetful<T,31><<<grid, block>>>(input,output,widthImage,heightImage);
            break;
	case 5:
        histogram2d<<<total_blocks, total_threads>>>(input,output,d_histogram,heightImage,widthImage,window_size);
            break; 
        }
        break;
    default:
        printf("That window size has not been implemented yet \n");
        //hipDeviceSynchronize();
    }
}

void MedianFilterUcharCUDA(unsigned char *input,unsigned char* output,unsigned int * d_histogram,int widthImage, int heightImage,int window_size, int threadsX, int threadsY,int implentation)
{
    callMedianFilter<unsigned char>(input,output,d_histogram,widthImage,heightImage,window_size, threadsX, threadsY,implentation);
}